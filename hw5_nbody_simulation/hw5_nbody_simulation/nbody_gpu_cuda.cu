#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <GL/glut.h>
#include <iostream>
#include <cmath>
#include <cstdlib>
#include <cstring>

using namespace std;

// Maximum block size
const int MAX_THREADS_PER_BLOCK = 1024;

// Simulation parameters
int n_bodies = 1000;       
float time_step = 0.01f;  
float damping = 0.99f;     
float softening_sq = 0.1f; 
int block_size = 1024;    

// Device pointers
float *d_x, *d_y, *d_z;       
float *d_mass;              
float *d_x_next, *d_y_next, *d_z_next; 
float *d_x_velocity, *d_y_velocity, *d_z_velocity; 
float *d_x_acceleration, *d_y_acceleration, *d_z_acceleration;

// Host arrays
float *x, *y, *z;
float *mass;
int window_width = 800;
int window_height = 800;

// CUDA kernel to compute accelerations
__global__ void computeAccelerations(
    int n_bodies, float *x, float *y, float *z, float *mass,
    float *x_acc, float *y_acc, float *z_acc, float softening_sq) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_bodies) return;

    float acc_x = 0.0f, acc_y = 0.0f, acc_z = 0.0f;

    for (int j = 0; j < n_bodies; j++) {
        if (i == j) continue;

        float dist_x = x[j] - x[i];
        float dist_y = y[j] - y[i];
        float dist_z = z[j] - z[i];

        float dist_sqr = dist_x * dist_x + dist_y * dist_y + dist_z * dist_z + softening_sq;
        float inv_dist_cube = rsqrtf(dist_sqr * dist_sqr * dist_sqr);

        // Reduce gravitational strength for very close distances
        if (dist_sqr < 0.01f) inv_dist_cube *= 0.1f;

        acc_x += dist_x * mass[j] * inv_dist_cube;
        acc_y += dist_y * mass[j] * inv_dist_cube;
        acc_z += dist_z * mass[j] * inv_dist_cube;
    }

    x_acc[i] = acc_x;
    y_acc[i] = acc_y;
    z_acc[i] = acc_z;
}

// CUDA kernel to update positions and velocities
__global__ void updatePositionsAndVelocities(
    int n_bodies, float time_step, float damping,
    float *x, float *y, float *z,
    float *x_next, float *y_next, float *z_next,
    float *vx, float *vy, float *vz,
    float *ax, float *ay, float *az, float *mass) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_bodies) return;

    vx[i] = (vx[i] + (ax[i] / mass[i]) * time_step) * damping;
    vy[i] = (vy[i] + (ay[i] / mass[i]) * time_step) * damping;
    vz[i] = (vz[i] + (az[i] / mass[i]) * time_step) * damping;

    x_next[i] = x[i] + vx[i] * time_step;
    y_next[i] = y[i] + vy[i] * time_step;
    z_next[i] = z[i] + vz[i] * time_step;
}

// OpenGL display function
void display() {
    int num_blocks = (n_bodies + block_size - 1) / block_size;

    // Launch kernel to compute accelerations
    computeAccelerations<<<num_blocks, block_size>>>(
        n_bodies, d_x, d_y, d_z, d_mass, d_x_acceleration, d_y_acceleration, d_z_acceleration, softening_sq);

    // Launch kernel to update positions and velocities
    updatePositionsAndVelocities<<<num_blocks, block_size>>>(
        n_bodies, time_step, damping, d_x, d_y, d_z, d_x_next, d_y_next, d_z_next,
        d_x_velocity, d_y_velocity, d_z_velocity, d_x_acceleration, d_y_acceleration, d_z_acceleration, d_mass);

    // Swap device arrays for the next iteration
    swap(d_x, d_x_next);
    swap(d_y, d_y_next);
    swap(d_z, d_z_next);

    // Copy updated positions back to host for rendering
    hipMemcpy(x, d_x, n_bodies * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(y, d_y, n_bodies * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(z, d_z, n_bodies * sizeof(float), hipMemcpyDeviceToHost);

    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    glBegin(GL_POINTS);
    for (int i = 0; i < n_bodies; i++) {
        glVertex3f(x[i], y[i], z[i]);
    }
    glEnd();

    glFlush();
    glutSwapBuffers();
    glutPostRedisplay();
}

// command-line arguments
void parseArguments(int argc, char **argv) {
    bool block_size_defined = false;

    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "--window_size") == 0 && i + 2 < argc) {
            window_width = atoi(argv[++i]);
            window_height = atoi(argv[++i]);
        } else if (strcmp(argv[i], "--n_bodies") == 0 && i + 1 < argc) {
            n_bodies = atoi(argv[++i]);
        } else if (strcmp(argv[i], "--time_step") == 0 && i + 1 < argc) {
            time_step = atof(argv[++i]);
        } else if (strcmp(argv[i], "--block_size") == 0 && i + 1 < argc) {
            block_size = atoi(argv[++i]);
            block_size_defined = true;
            if (block_size > MAX_THREADS_PER_BLOCK) {
                cerr << "Error: Block size cannot exceed " << MAX_THREADS_PER_BLOCK << "." << endl;
                exit(1);
            }
        }
    }

    if (!block_size_defined) {
        block_size = MAX_THREADS_PER_BLOCK;
        if (n_bodies > block_size) {
            cerr << "Error: Number of bodies exceeds the maximum supported block size of 1024." << endl;
            exit(1);
        }
    }
}


int main(int argc, char **argv) {
    parseArguments(argc, argv);
    // Initialize bodies with random positions and velocities
    // Initialize host arrays
    mass = new float[n_bodies];
    x = new float[n_bodies];
    y = new float[n_bodies];
    z = new float[n_bodies];
    float *vx = new float[n_bodies];
    float *vy = new float[n_bodies];
    float *vz = new float[n_bodies];

    for (int i = 0; i < n_bodies; i++) {
        mass[i] = 1.0f;
        x[i] = static_cast<float>(rand()) / RAND_MAX - 0.5f; 
        y[i] = static_cast<float>(rand()) / RAND_MAX - 0.5f;
        z[i] = static_cast<float>(rand()) / RAND_MAX - 0.5f;

        vx[i] = (static_cast<float>(rand()) / RAND_MAX - 0.5f) * 2.0f;
        vy[i] = (static_cast<float>(rand()) / RAND_MAX - 0.5f) * 2.0f;
        vz[i] = (static_cast<float>(rand()) / RAND_MAX - 0.5f) * 2.0f;
    }

    // Allocate device memory
    hipMalloc(&d_mass, n_bodies * sizeof(float));
    hipMalloc(&d_x, n_bodies * sizeof(float));
    hipMalloc(&d_y, n_bodies * sizeof(float));
    hipMalloc(&d_z, n_bodies * sizeof(float));
    hipMalloc(&d_x_next, n_bodies * sizeof(float));
    hipMalloc(&d_y_next, n_bodies * sizeof(float));
    hipMalloc(&d_z_next, n_bodies * sizeof(float));
    hipMalloc(&d_x_velocity, n_bodies * sizeof(float));
    hipMalloc(&d_y_velocity, n_bodies * sizeof(float));
    hipMalloc(&d_z_velocity, n_bodies * sizeof(float));
    hipMalloc(&d_x_acceleration, n_bodies * sizeof(float));
    hipMalloc(&d_y_acceleration, n_bodies * sizeof(float));
    hipMalloc(&d_z_acceleration, n_bodies * sizeof(float));

    // Copy data to device
    hipMemcpy(d_mass, mass, n_bodies * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, n_bodies * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n_bodies * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_z, z, n_bodies * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x_velocity, vx, n_bodies * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y_velocity, vy, n_bodies * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_z_velocity, vz, n_bodies * sizeof(float), hipMemcpyHostToDevice);

    // OpenGL setup
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE | GLUT_DEPTH);
    glutInitWindowSize(window_width, window_height);
    glutCreateWindow("CUDA N-Body Simulation");
    glutDisplayFunc(display);

    glEnable(GL_DEPTH_TEST);
    glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
    glPointSize(2.0f);

    glutMainLoop();

    // Cleanup
    delete[] mass;
    delete[] x;
    delete[] y;
    delete[] z;
    delete[] vx;
    delete[] vy;
    delete[] vz;

    hipFree(d_mass);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    hipFree(d_x_next);
    hipFree(d_y_next);
    hipFree(d_z_next);
    hipFree(d_x_velocity);
    hipFree(d_y_velocity);
    hipFree(d_z_velocity);
    hipFree(d_x_acceleration);
    hipFree(d_y_acceleration);
    hipFree(d_z_acceleration);

    return 0;
}

