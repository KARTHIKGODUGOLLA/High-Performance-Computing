#include "hip/hip_runtime.h"
#include <cstdint>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h> //for printf in cuda kernels


#include <iostream>
using std::cerr;
using std::cout;
using std::endl;

#include "device.hxx"
#include "exceptions.hxx"
#include "tensor.cuh"
#include "convolution_2d.cuh"


__global__ void gpu__convolve_2d(
    int16_t n_convolution_weights,
    int16_t out_channels,
    int16_t in_batch_width,
    int16_t in_channel_width,
    int16_t in_y_size,
    int16_t in_x_size,
    int16_t out_batch_width,
    int16_t out_channel_width,
    int16_t out_y_size,
    int16_t out_x_size,
    int16_t conv_y_size,
    int16_t conv_x_size,
    int16_t padding,
    float *input_values,
    float *output_values,
    float *weights
) {
    int16_t input_channel = blockIdx.x;
    int16_t batch = threadIdx.x;
    int16_t output_channel = threadIdx.y;
    //printf("running kernel for input channel: %d, batch: %d and output channel: %d\n", input_channel, batch, output_channel);

    // the only way to pass in a dynamic shared memory is to use
    // an extern __shared__ array like this
    extern __shared__ float shared_memory[];

    //get pointers to the N values in shared memory (used for the weights) for this input to
    //output convolution
    //and the next M values in shared memory (used for the input channel)
    float *convolve_shared_weights = shared_memory;
    float *shared_input_values = shared_memory + (out_channels * n_convolution_weights);

     // Load weights into shared memory
    for (int i = batch; i < n_convolution_weights; i += blockDim.x) {
        convolve_shared_weights[output_channel * n_convolution_weights + i] =
            weights[input_channel * out_channels * n_convolution_weights +
                    output_channel * n_convolution_weights + i];
    }

    // Load input values into shared memory
    for (int i = threadIdx.x; i < in_channel_width; i += blockDim.x) {
        shared_input_values[batch * in_channel_width + i] =
            input_values[batch * in_batch_width + input_channel * in_channel_width + i];
    }

    //make sure you sync the threads after loading the shared memory
    __syncthreads();


    //TODO: do the convolution operation here for the given batch, input channel and output channel
    //when summing up to the output channel make sure you use CUDA's atomicAdd function to prevent
    //race conditions when adding to the output channel's values
     for (int out_y = 0; out_y < out_y_size; ++out_y) {
        for (int out_x = 0; out_x < out_x_size; ++out_x) {
            float sum = 0.0f;

            for (int conv_y = 0; conv_y < conv_y_size; ++conv_y) {
                for (int conv_x = 0; conv_x < conv_x_size; ++conv_x) {
                    int in_y = out_y + conv_y - padding;
                    int in_x = out_x + conv_x - padding;

                    // Ensure input coordinates are valid
                    if (in_y >= 0 && in_y < in_y_size && in_x >= 0 && in_x < in_x_size) {
                        int input_idx = batch * in_channel_width + in_y * in_x_size + in_x;
                        int weight_idx = output_channel * n_convolution_weights +
                                         conv_y * conv_x_size + conv_x;

                        sum += shared_input_values[input_idx] * convolve_shared_weights[weight_idx];
                    }
                }
            }
            int output_idx = batch * out_batch_width + output_channel * out_channel_width +
                             out_y * out_x_size + out_x;
            atomicAdd(&output_values[output_idx], sum);

        }
    }
}


__global__ void gpu__convolve_2d_slow(
    int16_t n_convolution_weights,
    int16_t out_channels,
    int16_t in_batch_width,
    int16_t in_channel_width,
    int16_t in_y_size,
    int16_t in_x_size,
    int16_t out_batch_width,
    int16_t out_channel_width,
    int16_t out_y_size,
    int16_t out_x_size,
    int16_t conv_y_size,
    int16_t conv_x_size,
    int16_t padding,
    float *input_values,
    float *output_values,
    float *weights
) {
    int16_t input_channel = blockIdx.x;
    int16_t batch = threadIdx.x;
    int16_t output_channel = threadIdx.y;
    //printf("running kernel for input channel: %d, batch: %d and output channel: %d\n", input_channel, batch, output_channel);

    //TODO: implement the convolution operation just using global memory
    //when summing up to the output channel make sure you use CUDA's atomicAdd function to prevent
    //race conditions when adding to the output channel's values
    //convolution operation
    for (int16_t out_y = 0; out_y < out_y_size; ++out_y) {
        for (int16_t out_x = 0; out_x < out_x_size; ++out_x) {
            float sum = 0.0;

            for (int16_t conv_y = 0; conv_y < conv_y_size; ++conv_y) {
                for (int16_t conv_x = 0; conv_x < conv_x_size; ++conv_x) {
                    int16_t in_y = out_y + conv_y - padding;
                    int16_t in_x = out_x + conv_x - padding;

                    if (in_y >= 0 && in_y < in_y_size && in_x >= 0 && in_x < in_x_size) {
                        int input_idx = (batch * in_batch_width) +
                                        (input_channel * in_channel_width) +
                                        (in_y * in_x_size) + in_x;

                        int weight_idx = (input_channel * out_channels * n_convolution_weights) +
                                         (output_channel * n_convolution_weights) +
                                         (conv_y * conv_x_size) + conv_x;

                        sum += input_values[input_idx] * weights[weight_idx];
		    }
		}
	    }
            int output_idx = (batch * out_batch_width) +
                             (output_channel * out_channel_width) +
                             (out_y * out_x_size) + out_x;

            atomicAdd(&output_values[output_idx], sum);
        }
    }
}



Convolution2D::Convolution2D(const Device &_device, uint32_t _y_size, uint32_t _x_size, uint32_t _padding, Tensor *_input_tensor, Tensor *_output_tensor) : device(_device), y_size(_y_size), x_size(_x_size), padding(_padding), input_tensor(_input_tensor), output_tensor(_output_tensor) {

    //first verify that the size of the convolution and padding will appropriately work
    //between the input and output tensors

    int32_t expected_output_x = (input_tensor->x_size - x_size) + 1 + (padding * 2);
    int32_t expected_output_y = (input_tensor->y_size - y_size) + 1 + (padding * 2);

    if (expected_output_x != output_tensor->x_size || expected_output_y != output_tensor->y_size) {
        throw INVALID_CONVOLUTION_SIZE;
    }

    if (input_tensor->batch_size != output_tensor->batch_size) {
        throw BATCH_SIZE_MISMATCH;
    }

    n_convolution_weights = x_size * y_size;
    n_weights = n_convolution_weights * input_tensor->channels * output_tensor->channels;

    //grab these once to save on pointer dereferences
    batch_size = input_tensor->batch_size; //both tensors should have the same batch size
    in_channels = input_tensor->channels;
    in_y_size = input_tensor->y_size;
    in_x_size = input_tensor->x_size;

    out_channels = output_tensor->channels;
    out_y_size = output_tensor->y_size;
    out_x_size = output_tensor->x_size;

    //precompute widths so we can re-use these without extra compute
    in_batch_width = in_channels * in_y_size * in_x_size;
    in_channel_width = in_y_size * in_x_size;

    out_batch_width = out_channels * out_y_size * out_x_size;
    out_channel_width = out_y_size * out_x_size;

    if (device.is_cpu()) {
        weights = new float[n_weights];
        errors = new float[n_weights];
    } else {
        //allocate the memory on the GPU
        hipSetDevice(device.get_gpu());
        hipMallocManaged(&weights, n_weights * sizeof(float));
        hipMallocManaged(&errors, n_weights * sizeof(float));
    }
}


Convolution2D::~Convolution2D() {
    if (device.is_cpu()) {
        delete weights;
        delete errors;
    } else {
        hipFree(weights);
        hipFree(errors);
    }
}

void Convolution2D::forward() {
    if (device.is_cpu()) {
        uint32_t weight_offset = 0;

        float sum;

        int32_t conv_weight_offset; //used to go over the same set of weights for a convolution repeatedly
        int32_t current_in_x;
        int32_t current_in_y;

        //TODO: implement the convolution *WITHOUT* using the get/set/accumulate methods

        //iterate over every pair of input to output channels, there will be one
        //convolution between each. i.e., the input channels are fully connected
        //to the output channels.
        for (uint32_t input_channel = 0; input_channel < in_channels; input_channel++) {
            for (uint32_t output_channel = 0; output_channel < out_channels; output_channel++) {

                //instead of doing the batch first, each input to output channel will
                //use the same weights across the batch, so we can keep them in memory
                //better by iterating by batch after determining the input to output
                //channel
                for (uint32_t batch = 0; batch < batch_size; batch++) {
                    //TODO: calculate input and output offsets for the input and output channel values
                    uint32_t input_offset = batch * in_batch_width + input_channel * in_channel_width;
                    uint32_t output_offset = batch * out_batch_width + output_channel * out_channel_width;
                    //iterate over the input tensor's channel, we use the output
                    //sizes because if the input tensor is larger than the output
                    //the padding and convolution will make sure we hit every
                    //input cell
                    for (int32_t out_y = 0; out_y < out_y_size; out_y++) {
                        for (int32_t out_x = 0; out_x < out_x_size; out_x++) {

                            //TODO: do the convolution here
                            float sum = 0.0;
                            for (int32_t conv_y = 0; conv_y < y_size; conv_y++) {
                                for (int32_t conv_x = 0; conv_x < x_size; conv_x++) {
                                    // Compute input coordinates considering padding
                                    int32_t in_y = out_y + conv_y - padding;
                                    int32_t in_x = out_x + conv_x - padding;
                                    // make sure the input x and y values are actually inside the tensor                                    
                                    if (in_y >= 0 && in_y < in_y_size && in_x >= 0 && in_x < in_x_size) {
                                        float input_value = input_tensor->values[input_offset + in_y * in_x_size + in_x];

                                        uint32_t weight_index = (input_channel * out_channels + output_channel) * n_convolution_weights +
                                                                conv_y * x_size + conv_x;
                                        float weight_value = weights[weight_index];
                                        sum += input_value * weight_value;
                                    }
                                }
                            }
                            // Add the calculated sum to the output tensor's initial value
                            output_tensor->values[output_offset + out_y * out_x_size + out_x] += sum;
                        }
                    }

                }
            }
        }

    } else {
        //launch the CUDA kernel to do the forward pass using
        //shared memory

        //we will have one block per input channel
        int32_t number_blocks = in_channels;
        //in each block we will have out_channels (y dimension) threads per batch (x dimension)
        dim3 threads_per_block(batch_size, out_channels);
        //this will let each block share the weights across the batch for the convolution to each 
        //output channel for the input channel, and also have each block share the input channel's
        //values

        //this will allocate the shared memory that we will use to save the weights and each input channel
        //we will have the convolution weights for each output channel plus the input channel values for
        //the whole batch.
        int32_t shared_memory_size = ((n_convolution_weights * out_channels) + (batch_size * input_tensor->channel_width)) * sizeof(float);

        cout << "starting FAST GPU 2D convolution with number blocks: " << number_blocks << " and threads per block.x: " << threads_per_block.x << ", y: " << threads_per_block.y << " and z: " << threads_per_block.z << endl;
        cout << "batch_size: " << batch_size << ", out_channels: " << out_channels << ", shared memory size (n floats): " << (shared_memory_size / sizeof(float)) << endl;

        gpu__convolve_2d<<<number_blocks, threads_per_block, shared_memory_size>>>(
            n_convolution_weights,
            out_channels,
            in_batch_width,
            in_channel_width,
            in_y_size,
            in_x_size,
            out_batch_width,
            out_channel_width,
            out_y_size,
            out_x_size,
            y_size,
            x_size,
            padding,
            input_tensor->values,
            output_tensor->values,
            weights
        );
        hipDeviceSynchronize();

        hipError_t code = hipGetLastError();
        if (code != hipSuccess) {   
            cerr << "GPUassert: " << hipGetErrorString(code) << endl;
            exit(1);
        }   

    }
}

void Convolution2D::forward_slow() {
    if (device.is_cpu()) {
        float sum;

        int32_t current_in_x;
        int32_t current_in_y;

        //iterate over every pair of input to output channels, there will be one
        //convolution between each. i.e., the input channels are fully connected
        //to the output channels.
        for (uint32_t input_channel = 0; input_channel < in_channels; input_channel++) {
            for (uint32_t output_channel = 0; output_channel < out_channels; output_channel++) {

                //instead of doing the batch first, each input to output channel will
                //use the same weights across the batch, so we can keep them in memory
                //better by iterating by batch after determining the input to output
                //channel
                for (uint32_t batch = 0; batch < batch_size; batch++) {
                    //iterate over the input tensor's channel, we use the output
                    //sizes because if the input tensor is larger than the output
                    //the padding and convolution will make sure we hit every
                    //input cell
                    for (int32_t in_y = 0; in_y < out_y_size; in_y++) {
                        for (int32_t in_x = 0; in_x < out_x_size; in_x++) {

                            //reset the weight offset so we can
                            //iterate over the same set of weights
                            //for each convolution
                            sum = 0.0;
                            for (int32_t conv_y = 0; conv_y < y_size; conv_y++) {
                                for (int32_t conv_x = 0; conv_x < x_size; conv_x++) {
                                    current_in_y = in_y + conv_y - padding;
                                    current_in_x = in_x + conv_x - padding;
                                    
                                    // make sure the input x and y values are actually inside
                                    // the tensor, if they are outside they refer to a padding
                                    // cell
                                    if (current_in_y >= 0 && current_in_y < in_y_size && current_in_x >= 0 && current_in_x < in_x_size) {
                                        sum += input_tensor->get(batch, input_channel, current_in_y, current_in_x) * get(input_channel, output_channel, conv_y, conv_x);
                                    }
                                }
                            }

                            //cout << "setting output_tensor->values[" << batch << "][" << output_channel << "][" << in_y << "][" << in_x << "] += " << sum << endl;

                            //the starting x and y indexes of the input tensor are where
                            //we'll set the output tensor values
                            output_tensor->accumulate(batch, output_channel, in_y, in_x, sum);
                        }
                    }

                }
            }
        }
    } else {
        // run the GPU kernel without using shared memory

        //we will have one block per input channel
        int32_t number_blocks = in_channels;
        //in each block we will have in_channels (y dimension) threads per batch (x dimension)
        dim3 threads_per_block(batch_size, out_channels);

        cout << "starting SLOW GPU 2D convolution with number blocks: " << number_blocks << " and threads per block.x: " << threads_per_block.x << ", y: " << threads_per_block.y << " and z: " << threads_per_block.z << endl;

        cout << "batch_size: " << batch_size << ", out_channels: " << out_channels << endl;

        gpu__convolve_2d_slow<<<number_blocks, threads_per_block>>>(
            n_convolution_weights,
            out_channels,
            in_batch_width,
            in_channel_width,
            in_y_size,
            in_x_size,
            out_batch_width,
            out_channel_width,
            out_y_size,
            out_x_size,
            y_size,
            x_size,
            padding,
            input_tensor->values,
            output_tensor->values,
            weights
        );
        hipDeviceSynchronize();

        hipError_t code = hipGetLastError();
        if (code != hipSuccess) {   
            cerr << "GPUassert: " << hipGetErrorString(code) << endl;
            exit(1);
        }   

    }
}


float Convolution2D::get(int32_t in_channel, int32_t out_channel, int32_t y, int32_t x) {
    //there will be out_channels convolutions, each with n_convolution_weights weights
    //for every input channel.
    int32_t channel_width = out_channels * n_convolution_weights;

    return weights[(in_channel * channel_width) + (out_channel * n_convolution_weights) + (y * x_size) + x];
}

void Convolution2D::set(int32_t in_channel, int32_t out_channel, int32_t y, int32_t x, float value) {
    //there will be out_channels convolutions, each with n_convolution_weights weights
    //for every input channel.
    int32_t channel_width = out_channels * n_convolution_weights;

    weights[(in_channel * channel_width) + (out_channel * n_convolution_weights) + (y * x_size) + x] = value;
}


void Convolution2D::backward() {
    if (device.is_cpu()) {

    } else {
        //launch the CUDA kernel to do the backward pass

    }
}

void Convolution2D::reset() {
    if (device.is_cpu()) {
        memset(errors, 0, n_weights * sizeof(float));
    } else {
        hipMemset(errors, 0, n_weights * sizeof(float));
    }
}

void Convolution2D::print_weights() {
    int32_t counter = 0;
    for (int32_t in_channel = 0; in_channel < in_channels; in_channel++) {
        cout << "INPUT CHANNEL: " << in_channel << endl;
        for (int32_t out_channel = 0; out_channel < out_channels; out_channel++) {
            cout << "\tINPUT CHANNEL:" << in_channel << " to OUTPUT CHANNEL: " << out_channel << endl;
            for (int32_t y = 0; y < y_size; y++) {
                cout << "\t\t[";
                for (int32_t x = 0; x < x_size; x++) {
                    float value1 = get(in_channel, out_channel, y, x);
                    float value2 = weights[counter];

                    if (value1 != value2) {
                        cerr << "\n\nERROR: get and increment did not return same weight!" << endl;
                        exit(1);
                    }

                    cout << " " << value2;
                    counter++;
                }
                cout << " ]" << endl;
            }
        }
    }
    cout << endl;
}


ostream& operator<<(ostream& stream, const Convolution2D& convolution) {
    stream << "[convolution " << convolution.x_size << " x " << convolution.y_size << " | input channels: " << convolution.input_tensor->channels << ", output channels: "  << convolution.output_tensor->channels << ", n weights: " << convolution.n_weights << "]";

    return stream;
}

ostream& operator<<(ostream& stream, const Convolution2D *convolution) {
    // this will call the non-pointer version by dereferencing
    // the pointer to convolution
    stream << *convolution;
    return stream;
}

