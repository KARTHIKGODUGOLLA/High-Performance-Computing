#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <chrono>
using std::chrono::high_resolution_clock;
using std::chrono::duration_cast;
using std::chrono::duration;
using std::chrono::milliseconds;


#include <iostream>
using std::cerr;
using std::cout;
using std::endl;

#include <string>
using std::string;

#include "device.hxx"
#include "exceptions.hxx"
#include "tensor.cuh"
#include "convolution_2d.cuh"
#include "test_common.hxx"

bool tensors_close_enough(string t1_name, Tensor *t1, string t2_name, Tensor *t2) {
    float t1_value;
    float t2_value;

    bool mismatch = false;

    for (int32_t batch = 0; batch < t1->batch_size; batch++) {
        for (int32_t channel = 0; channel < t1->channels; channel++) {
            for (int32_t y = 0; y < t1->y_size; y++) {
                for (int32_t x = 0; x < t1->x_size; x++) {
                    t1_value = t1->get(batch, channel, y, x);
                    t2_value = t2->get(batch, channel, y, x);

                    if (!(close_enough(t1_value, t2_value))) {
                        cout << "tensors not close enough on batch: " << batch << ", channel: " << channel << ", y: " << y << ", x: " << x << ", " << t1_name << " value: " << t1_value << ", " << t2_name << " value: " << t2_value << endl;
                        mismatch = true;
                    }
                }
            }
        }
    }

    return !mismatch;
}

bool convolutions_close_enough(string c1_name, Convolution2D *c1, string c2_name, Convolution2D *c2) {
    float c1_value;
    float c2_value;

    bool mismatch = false;

    for (int32_t in_channel = 0; in_channel < c1->in_channels; in_channel++) {
        for (int32_t out_channel = 0; out_channel < c1->out_channels; out_channel++) {
            for (int32_t y = 0; y < c1->y_size; y++) {
                for (int32_t x = 0; x < c1->x_size; x++) {
                    c1_value = c1->get(in_channel, out_channel, y, x);
                    c2_value = c2->get(in_channel, out_channel, y, x);

                    if (!(close_enough(c1_value, c2_value))) {
                        cout << "convolutions not close enough on in channel: " << in_channel << ", out_channel: " << out_channel << ", y: " << y << ", x: " << x << ", " << c1_name << " value: " << c1_value << ", " << c2_name << " value: " << c2_value << endl;
                        mismatch = true;
                    }
                }
            }
        }
    }

    return !mismatch;
}


void randomize_tensor(Tensor *t) {
    for (int32_t batch = 0; batch < t->batch_size; batch++) {
        for (int32_t channel = 0; channel < t->channels; channel++) {
            for (int32_t y = 0; y < t->y_size; y++) {
                for (int32_t x = 0; x < t->x_size; x++) {
                    t->set(batch, channel, y, x, (drand48() * 2.0) - 1.0);
                }
            }
        }
    }
}

void randomize_convolution(Convolution2D *c) {
    for (int32_t in_channel = 0; in_channel < c->in_channels; in_channel++) {
        for (int32_t out_channel = 0; out_channel < c->out_channels; out_channel++) {
            for (int32_t y = 0; y < c->y_size; y++) {
                for (int32_t x = 0; x < c->x_size; x++) {
                    c->set(in_channel, out_channel, y, x, (drand48() * 2.0) - 1.0);
                }
            }
        }
    }
}

/**
 * Creates a copy of the tensor on the given device.
 */
Tensor* copy_tensor(const Device &device, Tensor *t) {
    Tensor *copy = new Tensor(device, t->batch_size, t->channels, t->y_size, t->x_size);

    for (int32_t i = 0; i < t->size; i++) {
        copy->values[i] = t->values[i];
    }

    return copy;
}

/**
 * Copies weights from one c1 into to c2.
 */
void copy_weights(Convolution2D *c1, Convolution2D *c2) {
    for (int32_t i = 0; i < c1->n_weights; i++) {
        c2->weights[i] = c1->weights[i];
    }
}


void large_validation(int32_t batch_size, int32_t in_channels, int32_t in_size, int32_t out_channels, int32_t out_size, int32_t conv_size, int32_t padding) {
    Device cpu_device(-1);
    Device gpu_device(0);
    //Create initial tensors and do a slow forward pass through the convolution as a baseline.

    //these tensors should only have one value
    Tensor *input_tensor = new Tensor(cpu_device, batch_size, in_channels, in_size, in_size);
    Tensor *output_tensor = new Tensor(cpu_device, batch_size, out_channels, out_size, out_size);
    randomize_tensor(input_tensor);
    randomize_tensor(output_tensor);

    Tensor *fast_cpu_input_tensor = copy_tensor(cpu_device, input_tensor);
    Tensor *slow_gpu_input_tensor = copy_tensor(gpu_device, input_tensor);
    Tensor *fast_gpu_input_tensor = copy_tensor(gpu_device, input_tensor);

    Tensor *fast_cpu_output_tensor = copy_tensor(cpu_device, output_tensor);
    Tensor *slow_gpu_output_tensor = copy_tensor(gpu_device, output_tensor);
    Tensor *fast_gpu_output_tensor = copy_tensor(gpu_device, output_tensor);

    Convolution2D *convolution = new Convolution2D(cpu_device, conv_size, conv_size, padding, input_tensor, output_tensor);
    randomize_convolution(convolution);

    Convolution2D *fast_cpu_convolution = new Convolution2D(cpu_device, conv_size, conv_size, padding, fast_cpu_input_tensor, fast_cpu_output_tensor);
    copy_weights(convolution, fast_cpu_convolution);

    Convolution2D *slow_gpu_convolution = new Convolution2D(gpu_device, conv_size, conv_size, padding, slow_gpu_input_tensor, slow_gpu_output_tensor);
    copy_weights(convolution, slow_gpu_convolution);

    Convolution2D *fast_gpu_convolution = new Convolution2D(gpu_device, conv_size, conv_size, padding, fast_gpu_input_tensor, fast_gpu_output_tensor);
    copy_weights(convolution, fast_gpu_convolution);

    auto slow_cpu_t1 = high_resolution_clock::now();
    convolution->forward_slow();
    auto slow_cpu_t2 = high_resolution_clock::now();

    hipDeviceSynchronize();
    auto fast_cpu_t1 = high_resolution_clock::now();
    fast_cpu_convolution->forward();
    auto fast_cpu_t2 = high_resolution_clock::now();

    auto slow_gpu_t1 = high_resolution_clock::now();
    slow_gpu_convolution->forward_slow();
    auto slow_gpu_t2 = high_resolution_clock::now();

    auto fast_gpu_t1 = high_resolution_clock::now();
    fast_gpu_convolution->forward();
    auto fast_gpu_t2 = high_resolution_clock::now();

    if (!tensors_close_enough("slow cpu", output_tensor, "fast cpu", fast_cpu_output_tensor)) {
        cout << "output tensor (slow cpu):" << endl;
        output_tensor->print_values();

        cout << "output tensor (fast cpu):" << endl;
        fast_cpu_output_tensor->print_values();

        cout << "output tensors (slow cpu vs fast cpu) were not close enough!" << endl;
        exit(1);
    }

    if (!tensors_close_enough("slow cpu", output_tensor, "slow gpu", slow_gpu_output_tensor)) {
        cout << "output tensor (slow cpu):" << endl;
        output_tensor->print_values();

        cout << "output tensor (slow gpu):" << endl;
        slow_gpu_output_tensor->print_values();

        cout << "output tensors (slow cpu vs slow gpu) were not close enough!" << endl;
        exit(1);
    }

    if (!tensors_close_enough("slow cpu", output_tensor, "fast gpu", fast_gpu_output_tensor)) {
        cout << "output tensor (slow cpu):" << endl;
        output_tensor->print_values();

        cout << "output tensor (fast gpu):" << endl;
        fast_gpu_output_tensor->print_values();

        cout << "output tensors (slow cpu vs fast gpu) were not close enough!" << endl;
        exit(1);
    }

    duration<double, std::milli> slow_cpu_time = slow_cpu_t2 - slow_cpu_t1;
    duration<double, std::milli> fast_cpu_time = fast_cpu_t2 - fast_cpu_t1;
    duration<double, std::milli> slow_gpu_time = slow_gpu_t2 - slow_gpu_t1;
    duration<double, std::milli> fast_gpu_time = fast_gpu_t2 - fast_gpu_t1;


    cout << "slow cpu runtime: " << slow_cpu_time.count() << " ms." << endl;
    cout << "fast cpu runtime: " << fast_cpu_time.count() << " ms." << endl;
    cout << "slow gpu runtime: " << slow_gpu_time.count() << " ms." << endl;
    cout << "fast gpu runtime: " << fast_gpu_time.count() << " ms." << endl;

    delete input_tensor;
    delete output_tensor;
    delete convolution;

    delete fast_cpu_input_tensor;
    delete fast_cpu_output_tensor;
    delete fast_cpu_convolution;

    delete slow_gpu_input_tensor;
    delete slow_gpu_output_tensor;
    delete slow_gpu_convolution;

    delete fast_gpu_input_tensor;
    delete fast_gpu_output_tensor;
    delete fast_gpu_convolution;
}

int main(int argc, char** argv) {
    int32_t batch_size = 5;
    int32_t in_channels = 8;
    int32_t in_size = 8;
    int32_t out_channels = 16;
    int32_t out_size = 8;
    int32_t conv_size = 3;
    int32_t padding = 1;

    try {
        large_validation(batch_size, in_channels, in_size, out_channels, out_size, conv_size, padding);
    } catch (int error) {
        cout << "ERROR was: " << ERROR_NAMES[error] << endl;
        exit(1);
    }

    batch_size = 20;
    in_channels = 16;
    in_size = 16;
    out_channels = 32;
    out_size = 16;
    conv_size = 3;
    padding = 1;

    try {
        large_validation(batch_size, in_channels, in_size, out_channels, out_size, conv_size, padding);
    } catch (int error) {
        cout << "ERROR was: " << ERROR_NAMES[error] << endl;
        exit(1);
    }

    return 0;
}

