#include <cstdint>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>
using std::cerr;
using std::cout;
using std::endl;

#include "device.hxx"
#include "tensor.cuh"


Tensor::Tensor(const Device &_device, uint32_t _batch_size, uint32_t _channels, uint32_t _y_size, uint32_t _x_size) : device(_device), batch_size(_batch_size), channels(_channels), y_size(_y_size), x_size(_x_size) {

    //get the width of each dimension in the 1D float array
    channel_width = y_size * x_size;
    batch_width = channels * channel_width;
    size = batch_size * batch_width;

    if (device.is_cpu()) {
        values = new float[size];
        errors = new float[size];
    } else {
        //allocate the memory on the GPU
        hipSetDevice(device.get_gpu());
        hipMallocManaged(&values, size * sizeof(float));
        hipMallocManaged(&errors, size * sizeof(float));
    }
}


Tensor::~Tensor() {
    if (device.is_cpu()) {
        delete values;
        delete errors;
    } else {
        hipFree(values);
        hipFree(errors);
    }
}

float Tensor::get(int32_t batch, int32_t channel, int32_t y, int32_t x) {
    return values[(batch * batch_width) + (channel * channel_width) + (y * x_size) + x];

}

void Tensor::set(int32_t batch, int32_t channel, int32_t y, int32_t x, float value) {
    values[(batch * batch_width) + (channel * channel_width) + (y * x_size) + x] = value;
}

void Tensor::accumulate(int32_t batch, int32_t channel, int32_t y, int32_t x, float value) {
    values[(batch * batch_width) + (channel * channel_width) + (y * x_size) + x] += value;
}



void Tensor::reset() {
    if (device.is_cpu()) {
        memset(values, 0, size * sizeof(float));
        memset(errors, 0, size * sizeof(float));
    } else {
        hipMemset(values, 0, size * sizeof(float));
        hipMemset(errors, 0, size * sizeof(float));
    }
}

void Tensor::print_values() {
    int32_t counter = 0;
    for (int32_t batch = 0; batch < batch_size; batch++) {
        cout << "BATCH: " << batch << endl;
        for (int32_t channel = 0; channel < channels; channel++) {
            cout << "\tBATCH " << batch << ", CHANNEL: " << channel << endl;
            for (int32_t y = 0; y < y_size; y++) {
                cout << "\t\t[";
                for (int32_t x = 0; x < x_size; x++) {
                    float value1 = get(batch, channel, y, x);
                    float value2 = values[counter];

                    if (value1 != value2) {
                        cerr << "\n\nERROR: get and increment did not return same value!" << endl;
                        exit(1);
                    }

                    cout << " " << value2;
                    counter++;
                }
                cout << " ]" << endl;
            }
        }
    }
    cout << endl;
}

ostream& operator<<(ostream& stream, const Tensor& tensor) {
    stream << "[tensor | " << tensor.batch_size << " x " << tensor.channels << " x " << tensor.y_size << " x " << tensor.x_size << "]";

    return stream;
}

ostream& operator<<(ostream& stream, const Tensor *tensor) {
    // this will call the non-pointer version by dereferencing
    // the pointer to tensor
    stream << *tensor;
    return stream;
}


